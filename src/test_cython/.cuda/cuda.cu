
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
using namespace std;

__global__ void 
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = alpha * x[index] + y[index];
}

bool saxpy_cuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    // allocate device memory buffers on the GPU using cudaMalloc
    hipMalloc((void **) &device_x, N * sizeof(float));
    hipMalloc((void **) &device_y, N * sizeof(float));
    hipMalloc((void **) &device_result, N * sizeof(float));

    // copy input arrays to the GPU using cudaMemcpy
    hipMemcpy(device_x, xarray, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, N * sizeof(float), hipMemcpyHostToDevice);

    // run kernel
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();

    // copy result from GPU using cudaMemcpy
    hipMemcpy(resultarray, device_result, N * sizeof(float), hipMemcpyDeviceToHost);

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        return false;
    }

    // free memory buffers on the GPU
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);

    return true;
}